
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include <assert.h>
 #include <fstream>
 #include <iostream>

 #include "hipsolver.h"
 #include "hipsolver.h"
 
 #include "helper_string.h"
 #include "helper_cusolver.h"
 
 #include "hipsolver.h"
 
 #include <hip/hip_runtime.h>
 #include "hip/hip_runtime_api.h"
 
#include "cudaSparseLinearSolvers.h"


#include <memory>
#include <cstring>


using namespace std;

void cudaRefactor::initialiseCudaHandles()
{
    checkCudaErrors(hipsolverSpCreate(&cusolverSpH));
    checkCudaErrors(hipsparseCreate(&cusparseH));
    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipStreamCreate(&stream2));


    checkCudaErrors(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
    checkCudaErrors(hipStreamCreateWithFlags(&stream2, hipStreamNonBlocking));



    checkCudaErrors(hipsolverSpSetStream(cusolverSpH, stream));   // Cusolver Stream and Handle link
    checkCudaErrors(hipsolverSpSetStream(cusolverSpH, stream2));   // Cusolver Stream and Handle link

    // checkCudaErrors(hipsparseSetStream(cusolverRfH, stream2));       // Cusparse stream and handle link


    checkCudaErrors(hipsparseCreateMatDescr(&descrA));
    checkCudaErrors(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));

    checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));

}


void cudaRefactor::destroyCudahandles()
{
    if (cusolverRfH) { checkCudaErrors(hipsolverRfDestroy(cusolverRfH)); }
    if (cusolverSpH) { checkCudaErrors(hipsolverSpDestroy(cusolverSpH)); }
    if (cusparseH  ) { checkCudaErrors(hipsparseDestroy(cusparseH)); }
    if (stream     ) { checkCudaErrors(hipStreamDestroy(stream)); }
    if (descrA     ) { checkCudaErrors(hipsparseDestroyMatDescr(descrA)); }
    if (info       ) { checkCudaErrors(cusolverSpDestroyCsrluInfoHost(info)); }
}


void cudaRefactor::LU_DecompositionHost(int* h_csrRowPtrA, int* h_csrColIndA, 
                                        double* h_csrValA, int _rowsA, int _nnzA,
                                         double* h_b , const char* reorder)
{
   rowsA = _rowsA;
   colsA = _rowsA;
   nnzA = _nnzA;
//    cout << " rows A : " << rowsA << " colsA : " << colsA << " nnz : " << nnzA <<endl;

   double* h_x;   // LOcal Declaration
    cout << " -------LU FACTOR ----------"  <<endl;

    /// -------- Memory Allocation for CPU arrays --------------- // 
    h_Qreorder   = (int*)malloc(sizeof(int)*colsA);
    h_csrColIndB = new int[nnzA]();
    h_csrRowPtrB = new int[rowsA+1]();
    h_csrValB    = (double*)malloc(sizeof(double)*nnzA);
    h_mapBfromA  = (int*   )malloc(sizeof(int)*nnzA);

    h_x    = (double*)malloc(sizeof(double)*colsA);
    // h_b    = (double*)malloc(sizeof(double)*rowsA);
    h_xhat = (double*)malloc(sizeof(double)*colsA);
    h_bhat = (double*)malloc(sizeof(double)*rowsA);

    assert(NULL != h_Qreorder); assert(NULL != h_csrRowPtrB); assert(NULL != h_csrColIndB);assert(NULL != h_csrValB);
    assert(NULL != h_mapBfromA);assert(NULL != h_x);assert(NULL != h_xhat);assert(NULL != h_bhat);

    // cout << " INFO :  Reordering The Matrix using " ;

    if ( 0 == strcmp(reorder, "symrcm") )
    {
        printf("  SYMRCM ==>  Q = symrcm(A)  \n" );
        checkCudaErrors(cusolverSpXcsrsymrcmHost(
            cusolverSpH, rowsA, nnzA,
            descrA, h_csrRowPtrA, h_csrColIndA, 
            h_Qreorder));
    }
    else if ( 0 == strcmp(reorder, "symamd") )
    {
        printf("   SYMAND ==>      Q = symamd(A)  \n" );
        checkCudaErrors(cusolverSpXcsrsymamdHost(
            cusolverSpH, rowsA, nnzA,
            descrA, h_csrRowPtrA, h_csrColIndA, 
            h_Qreorder));
    }

    else if ( 0 == strcmp(reorder, "metis") )
    {
        printf("    METIS ==>    Q = metis(A)  \n" );
        cusolverSpXcsrmetisndHost(
            cusolverSpH, rowsA, nnzA,
            descrA, h_csrRowPtrA, h_csrColIndA,NULL ,
            h_Qreorder);
    }
    else 
    {
        printf(" Error in Reordering \n" );
        // exit(0);
    }

    // printf("step 3: B = Q*A*Q^T\n");
    memcpy(h_csrRowPtrB, h_csrRowPtrA, sizeof(int)*(rowsA+1));
    memcpy(h_csrColIndB, h_csrColIndA, sizeof(int)*nnzA);

    // cout << " memcopy done " <<endl;
    // checkCudaErrors(hipDeviceSynchronize());

    start = second();
    start = second();

    checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(
        cusolverSpH, rowsA, colsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        h_Qreorder, h_Qreorder,
        &size_perm));

    if (buffer_cpu) {
        free(buffer_cpu); 
    }
    buffer_cpu = (void*)malloc(sizeof(char)*size_perm);
    assert(NULL != buffer_cpu);

    // h_mapBfromA = Identity 
    for(int j = 0 ; j < nnzA ; j++){
        h_mapBfromA[j] = j;
    }
    checkCudaErrors(cusolverSpXcsrpermHost(
        cusolverSpH, rowsA, colsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        h_Qreorder, h_Qreorder,
        h_mapBfromA,
        buffer_cpu));

    // B = A( mapBfromA )
    for(int j = 0 ; j < nnzA ; j++){
        h_csrValB[j] = h_csrValA[ h_mapBfromA[j] ];
    }

    stop = second();
    time_perm = stop - start;

    // printf("step 4: solve A*x = b by LU(B) in cusolverSp\n");

    // printf("step 4.1: create opaque info structure\n");
    checkCudaErrors(cusolverSpCreateCsrluInfoHost(&info));

    // printf("step 4.2: analyze LU(B) to know structure of Q and R, and upper bound for nnz(L+U)\n");
    start = second();
    start = second();

    checkCudaErrors(cusolverSpXcsrluAnalysisHost(
        cusolverSpH, rowsA, nnzA,
        descrA, h_csrRowPtrB, h_csrColIndB,
        info));

    stop = second();
    time_sp_analysis = stop - start;

    // printf("step 4.3: workspace for LU(B)\n");
    checkCudaErrors(cusolverSpDcsrluBufferInfoHost(
        cusolverSpH, rowsA, nnzA,
        descrA, h_csrValB, h_csrRowPtrB, h_csrColIndB,
        info,
        &size_internal,
        &size_lu));

    if (buffer_cpu) { 
        free(buffer_cpu); 
    }
    buffer_cpu = (void*)malloc(sizeof(char)*size_lu);
    assert(NULL != buffer_cpu);

    // printf("step 4.4: compute Ppivot*B = L*U \n");
    start = second();
    start = second();
    checkCudaErrors(cusolverSpDcsrluFactorHost(
        cusolverSpH, rowsA, nnzA,
        descrA, h_csrValB, h_csrRowPtrB, h_csrColIndB,
        info, pivot_threshold,
        buffer_cpu));

    stop = second();
    time_sp_factor = stop - start;

    // TODO: check singularity by tol
    // printf("step 4.5: check if the matrix is singular \n");
    checkCudaErrors(cusolverSpDcsrluZeroPivotHost(
        cusolverSpH, info, tol, &singularity));

    if ( 0 <= singularity)
    {
        fprintf(stderr, "Error: A is not invertible, singularity=%d\n", singularity);
        exit(0);
    }

    // printf("step 4.6: solve A*x = b \n");
    // printf("    i.e.  solve B*(Qx) = Q*b \n");
    start = second();
    start = second();

    // b_hat = Q*b
    for(int j = 0 ; j < rowsA ; j++){
        h_bhat[j] = h_b[h_Qreorder[j]];
    }
    // B*x_hat = b_hat
    checkCudaErrors(cusolverSpDcsrluSolveHost(
        cusolverSpH, rowsA, h_bhat, h_xhat, info, buffer_cpu));

    // x = Q^T * x_hat    // Solution Replaces B Vector
    for(int j = 0 ; j < rowsA ; j++){
        h_b[h_Qreorder[j]] = h_xhat[j];
    }

    stop = second();
    time_sp_solve = stop - start;

    // printf("step 5: extract P, Q, L and U from P*B*Q^T = L*U \n");
    // printf("        L has implicit unit diagonal\n");
    start = second();
    start = second();

    checkCudaErrors(cusolverSpXcsrluNnzHost(
        cusolverSpH,
        &nnzL,
        &nnzU,
        info));

    h_Plu = (int*)malloc(sizeof(int)*rowsA);
    h_Qlu = (int*)malloc(sizeof(int)*colsA);

    h_csrValL    = (double*)malloc(sizeof(double)*nnzL);
    h_csrRowPtrL = (int*)malloc(sizeof(int)*(rowsA+1)); 
    h_csrColIndL = (int*)malloc(sizeof(int)*nnzL);

    h_csrValU    = (double*)malloc(sizeof(double)*nnzU);
    h_csrRowPtrU = (int*)malloc(sizeof(int)*(rowsA+1)); 
    h_csrColIndU = (int*)malloc(sizeof(int)*nnzU);

    assert(NULL != h_Plu);
    assert(NULL != h_Qlu);

    assert(NULL != h_csrValL);
    assert(NULL != h_csrRowPtrL);
    assert(NULL != h_csrColIndL);

    assert(NULL != h_csrValU);
    assert(NULL != h_csrRowPtrU);
    assert(NULL != h_csrColIndU);

    checkCudaErrors(cusolverSpDcsrluExtractHost(
        cusolverSpH,
        h_Plu,
        h_Qlu,
        descrA,
        h_csrValL, 
        h_csrRowPtrL,
        h_csrColIndL,
        descrA,
        h_csrValU,
        h_csrRowPtrU,
        h_csrColIndU,
        info,
        buffer_cpu));

    stop = second();
    time_sp_extract = stop - start;

    // printf("nnzL = %d, nnzU = %d\n", nnzL, nnzU);
    

    printf("step 6: form P*A*Q^T = L*U\n");

    
    h_P = (int*)malloc(sizeof(int)*rowsA);
    h_Q = (int*)malloc(sizeof(int)*colsA);
    hipHostAlloc( (void**)&h_P, sizeof(int)*rowsA, hipHostMallocDefault) ;
    hipHostAlloc((void**)&h_Q, sizeof(int)*colsA, hipHostMallocDefault);
    // checkCudaErrors( hipHostRegister(h_P, sizeof(int)*rowsA, hipHostRegisterPortable) );
    // checkCudaErrors( hipHostRegister(h_Q, sizeof(int)*colsA, hipHostRegisterPortable) );


    assert(NULL != h_P);
    assert(NULL != h_Q);

    // printf("step 6.1: P = Plu*Qreroder\n");
    // gather operation, P = Qreorder(Plu)
    for(int j = 0 ; j < rowsA ; j++){
        h_P[j] = h_Qreorder[h_Plu[j]];
    }

    // printf("step 6.2: Q = Qlu*Qreorder \n");
    // gather operation, Q = Qreorder(Qlu)
    for(int j = 0 ; j < colsA ; j++){
        h_Q[j] = h_Qreorder[h_Qlu[j]];
    }

    delete[] h_x;


        printf("--- REFACTORIZE ");
    checkCudaErrors(hipsolverRfCreate(&cusolverRfH));



    printf("step 8: set parameters for cusolverRf \n");
    // numerical values for checking "zeros" and for boosting.
    checkCudaErrors(hipsolverRfSetNumericProperties(cusolverRfH, nzero, nboost));

    // choose algorithm for refactorization and solve
    checkCudaErrors(hipsolverRfSetAlgs(cusolverRfH, fact_alg, solve_alg));

    // matrix mode: L and U are CSR format, and L has implicit unit diagonal
    checkCudaErrors(hipsolverRfSetMatrixFormat(
        cusolverRfH, HIPSOLVERRF_MATRIX_FORMAT_CSR, HIPSOLVERRF_UNIT_DIAGONAL_ASSUMED_L));

    // fast mode for matrix assembling
    checkCudaErrors(hipsolverRfSetResetValuesFastMode(
        cusolverRfH, HIPSOLVERRF_RESET_VALUES_FAST_MODE_ON));
    // cout << " Rows a : "<< rowsA << " colsa : " << colsA << " nnzA : "<< nnzA <<endl; 

    checkCudaErrors(hipMalloc((void **)&d_csrRowPtrA, sizeof(int)*(rowsA+1)));
    checkCudaErrors(hipMalloc((void **)&d_csrColIndA, sizeof(int)*nnzA));
    checkCudaErrors(hipMalloc((void **)&d_csrValA   , sizeof(double)*nnzA));
    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(double)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_b, sizeof(double)*rowsA));
    checkCudaErrors(hipMalloc((void **)&d_P, sizeof(int)*rowsA));
    checkCudaErrors(hipMalloc((void **)&d_Q, sizeof(int)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_T, sizeof(double)*rowsA*1));
    // cout<< " Finished LU Host " <<endl;
}



void cudaRefactor::cudaRefactorize(int* h_csrRowPtrA, int* h_csrColIndA, double* h_csrValA, int rowsA, int N_iteration)
{


    checkCudaErrors(hipMemcpyAsync(d_csrRowPtrA, h_csrRowPtrA, sizeof(int)*(rowsA+1), hipMemcpyHostToDevice,stream));
    checkCudaErrors(hipMemcpyAsync(d_csrColIndA, h_csrColIndA, sizeof(int)*nnzA     , hipMemcpyHostToDevice,stream));
    checkCudaErrors(hipMemcpyAsync(d_csrValA   , h_csrValA   , sizeof(double)*nnzA  , hipMemcpyHostToDevice,stream));

            // Allocate Memory on Device
    if(N_iteration == 1)
    {
        checkCudaErrors(hipMemcpyAsync(d_P, h_P, sizeof(int)*rowsA, hipMemcpyHostToDevice,stream));
        checkCudaErrors(hipMemcpyAsync(d_Q, h_Q, sizeof(int)*colsA, hipMemcpyHostToDevice,stream));
        checkCudaErrors(hipsolverRfSetupHost(
                rowsA, nnzA, h_csrRowPtrA, h_csrColIndA, h_csrValA,nnzL, 
                h_csrRowPtrL, h_csrColIndL, h_csrValL, nnzU, h_csrRowPtrU, h_csrColIndU, h_csrValU, h_P, h_Q, 
                cusolverRfH));
        checkCudaErrors(hipsolverRfAnalyze(cusolverRfH));
    }

    start = second();
    // checkCudaErrors(hipStreamSynchronize(stream));
    
    checkCudaErrors(hipsolverRfResetValues(
        rowsA,nnzA,
        d_csrRowPtrA, d_csrColIndA, d_csrValA,
        d_P,
        d_Q,
        cusolverRfH));
    

    checkCudaErrors(hipsolverRfRefactor(cusolverRfH));

    // checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_rf_refactor = stop - start;
    printf("time_rf_refactor :  %f", time_rf_refactor);
    // checkCudaErrors(hipDeviceSynchronize());


    // Delete the Non used Device arrays

}


void cudaRefactor::cudaRefactorSolve(double* h_b)
{
    // cout << "----- Refactor and solve --- " <<endl;
    
    checkCudaErrors(hipMemcpyAsync(d_x, h_b, sizeof(double)*rowsA, hipMemcpyHostToDevice,stream));
    start = second();
    start = second();

    checkCudaErrors(hipStreamSynchronize(stream));
    
    checkCudaErrors(hipsolverRfSolve(cusolverRfH, d_P, d_Q, 1, d_T, rowsA, d_x, rowsA));

    // checkCudaErrors(hipDeviceSynchronize());
    stop = second();
    time_rf_solve = stop - start;

    checkCudaErrors(hipMemcpy(h_b, d_x, sizeof(double)*colsA, hipMemcpyDeviceToHost));

    // printf("===== timing profile \n");
    // // printf(" reorder A   : %f sec\n", time_reorder);
    // printf(" B = Q*A*Q^T : %f sec\n", time_perm);s
    // printf("\n");
    // printf(" cusolverSp LU analysis: %f sec\n", time_sp_analysis);
    // printf(" cusolverSp LU factor  : %f sec\n", time_sp_factor);
    // printf(" cusolverSp LU solve   : %f sec\n", time_sp_solve);
    // printf(" cusolverSp LU extract : %f sec\n", time_sp_extract);
    // printf("\n");
    // printf(" cusolverRf assemble : %f sec\n", time_rf_assemble);
    // printf(" cusolverRf reset    : %f sec\n", time_rf_reset);
    // printf(" cusolverRf refactor : %f sec\n", time_rf_refactor);
    // printf(" cusolverRf solve    : %f sec\n", time_rf_solve);



   
}


// Deletes all the LU factor Entities that was created as part of the Routines and calls back the Destroy handles routines
void cudaRefactor::resetCudaRF()
{
    // cout << " ------ REST CUDA RF " <<endl;

    // Delete all array created 
    if (h_Qreorder  ) { free(h_Qreorder); }
    
    if (h_csrRowPtrB) { free(h_csrRowPtrB); }
    if (h_csrColIndB) { free(h_csrColIndB); }
    if (h_csrValB   ) { free(h_csrValB   ); }
    if (h_mapBfromA ) { free(h_mapBfromA ); }

    if (h_xhat) { free(h_xhat); }
    if (h_bhat) { free(h_bhat); }

    // if (buffer_cpu) { free(buffer_cpu); }

    if (h_Plu) { free(h_Plu); }
    if (h_Qlu) { free(h_Qlu); }
    if (h_csrRowPtrL) { free(h_csrRowPtrL); }
    if (h_csrColIndL) { free(h_csrColIndL); }
    if (h_csrValL   ) { free(h_csrValL   ); }
    if (h_csrRowPtrU) { free(h_csrRowPtrU); }
    if (h_csrColIndU) { free(h_csrColIndU); }
    if (h_csrValU   ) { free(h_csrValU   ); }

    if (h_P) { checkCudaErrors(hipHostFree(h_P)); }
    if (h_Q) { checkCudaErrors(hipHostFree(h_Q)); }

    
    // if (h_P) { free(h_P); }
    // if (h_Q) { free(h_Q); }

    if (d_x) { checkCudaErrors(hipFree(d_x)); }
    if (d_P) { checkCudaErrors(hipFree(d_P)); }
    if (d_Q) { checkCudaErrors(hipFree(d_Q)); }
    if (d_T) { checkCudaErrors(hipFree(d_T)); }

    if (d_csrValA   ) { checkCudaErrors(hipFree(d_csrValA)); }
    if (d_csrRowPtrA) { checkCudaErrors(hipFree(d_csrRowPtrA)); }
    if (d_csrColIndA) { checkCudaErrors(hipFree(d_csrColIndA)); }
    if (d_b) { checkCudaErrors(hipFree(d_b)); }

    // Destroy ALl the handles
    destroyCudahandles();
}























































void cudaLowLevelQR::initialiseLowLevelCudaQR()
{

    checkCudaErrors(hipsolverSpCreate(&cusolverSpH));
    checkCudaErrors(hipsparseCreate(&cusparseH));
    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipsolverSpSetStream(cusolverSpH, stream));
    checkCudaErrors(hipsparseSetStream(cusparseH, stream));

    checkCudaErrors(hipsparseCreateMatDescr(&descrA));

    checkCudaErrors(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));

    if (baseA) 
    {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    }
    else
    {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
    }
}


void cudaLowLevelQR::lowLevelQRFactorisation(int* h_csrRowPtrA, int* h_csrColIndA, 
                                double* h_csrValA, int _rowsA,int _nnzA
                                 ,const char* reorder)

{

    rowsA = _rowsA;
    colsA = _rowsA;
    nnzA  = _nnzA;

    double tt1 , tt2;

    tt1 = second();
    // Allocate Memory on the Host
    h_x        = (double*)malloc(sizeof(double)*colsA);
    h_Qb       = (double*)malloc(sizeof(double)*rowsA);
   //  h_r        = (double*)malloc(sizeof(double)*rowsA);
    h_Q          = (int*   )malloc(sizeof(int)*colsA);
    h_csrRowPtrB = (int*   )malloc(sizeof(int)*(rowsA+1));
    h_csrColIndB = (int*   )malloc(sizeof(int)*nnzA);
    h_csrValB    = (double*)malloc(sizeof(double)*nnzA);
    h_mapBfromA  = (int*   )malloc(sizeof(int)*nnzA);

    tt2 = second();

    cout << " Malloc : " << tt2 - tt1  <<endl;


    assert(NULL != h_x);
    assert(NULL != h_csrRowPtrB );
    assert(NULL != h_csrColIndB );
    assert(NULL != h_csrValB    );
    assert(NULL != h_mapBfromA  );
    assert(NULL != h_Q  );
    



    //Allocate Memory on the Device 
     checkCudaErrors(hipMalloc((void **)&d_csrRowPtrB, sizeof(int)*(rowsA+1)));
     checkCudaErrors(hipMalloc((void **)&d_csrColIndB, sizeof(int)*nnzA));
     checkCudaErrors(hipMalloc((void **)&d_csrValB   , sizeof(double)*nnzA));
     checkCudaErrors(hipMalloc((void **)&d_Q, sizeof(int)*colsA));
     checkCudaErrors(hipMalloc((void **)&d_z, sizeof(double)*colsA));
     checkCudaErrors(hipMalloc((void **)&d_x, sizeof(double)*colsA));
     checkCudaErrors(hipMalloc((void **)&d_Qb, sizeof(double)*rowsA));

        // tt1 = second();
     // Reorder the Host Matrix
     if ( 0 == strcmp(reorder, "symrcm") )
     {
        printf(" SYMRCM Reordering Performed ");
         printf("step 2.1: Q = symrcm(A) \n");
         checkCudaErrors(cusolverSpXcsrsymrcmHost(
            cusolverSpH, rowsA, nnzA,
             descrA, h_csrRowPtrA,h_csrColIndA,
             h_Q));
     }
     else if ( 0 == strcmp(reorder, "symamd") )
     {
        printf(" SYMAND Reordering Performed ");
         printf("step 2.1: Q = symamd(A) \n");
         checkCudaErrors(cusolverSpXcsrsymamdHost(
            cusolverSpH, rowsA, nnzA,
             descrA,h_csrRowPtrA,h_csrColIndA,
             h_Q));
     }
     else if ( 0 == strcmp(reorder, "metis") )
     {
        //  printf("step 2.1: Q = metis(A) \n");
        checkCudaErrors(cusolverSpXcsrmetisndHost(
           cusolverSpH, rowsA, nnzA,
          descrA, h_csrRowPtrA, h_csrColIndA,
            NULL, /* default setting. */
            h_Q)); 

     }
     else 
     {
         fprintf(stderr, "Error: %s is unknown reordering\n", reorder);
         exit(0);
     }


    // Get a current copy of the matrix to array B
    memcpy(h_csrRowPtrB, h_csrRowPtrA, sizeof(int)*(rowsA+1));
    memcpy(h_csrColIndB, h_csrColIndA, sizeof(int)*nnzA);


     /* h_mapBfromA = Identity */
     for(int j = 0 ; j < nnzA ; j++)
     h_mapBfromA[j] = j;
 
    size_t size_perm;
    checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(
         cusolverSpH, rowsA, colsA, nnzA,
         descrA, h_csrRowPtrB, h_csrColIndB,
         h_Q, h_Q,
         &size_perm));
 
    //  if (buffer_cpu)
    //  {
    //      free(buffer_cpu);
    //  }
     buffer_cpu = (void*)malloc(sizeof(char)*size_perm);
     assert(NULL != buffer_cpu);
 
     // Compute Permutation 
     checkCudaErrors(cusolverSpXcsrpermHost(
         cusolverSpH, rowsA, colsA, nnzA,
         descrA, h_csrRowPtrB, h_csrColIndB,
         h_Q, h_Q,
         h_mapBfromA,
         buffer_cpu));
     
     // Calculate B = Q*A*Q'
     for(int j = 0 ; j < nnzA ; j++)
     {
         h_csrValB[j] = h_csrValA[ h_mapBfromA[j] ];
     }

     

    //  hipStreamSynchronize(stream);
    //  tt2 = second();
    //  cout << " PRe QR TIME TAKEN " << tt2 - tt1 <<endl;

    // Get a current copy of the matrix to array B
    checkCudaErrors(hipMemcpyAsync(d_csrRowPtrB, h_csrRowPtrB, sizeof(int)*(rowsA+1), hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_csrColIndB, h_csrColIndB, sizeof(int)*nnzA     , hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_csrValB   , h_csrValB   , sizeof(double)*nnzA  , hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_Q         , h_Q         , sizeof(int)*rowsA    , hipMemcpyHostToDevice, stream));



    // printf("step 2: create opaque info structure\n");
    checkCudaErrors(cusolverSpCreateCsrqrInfo(&d_info));

    // hipStreamSynchronize(stream); tt1 = second();
    // printf("step 3: analyze qr(A) to know structure of L\n");
    checkCudaErrors(cusolverSpXcsrqrAnalysis(
    cusolverSpH, rowsA, colsA, nnzA,
    descrA, d_csrRowPtrB, d_csrColIndB,
    d_info));
    // hipStreamSynchronize(stream); tt2 = second();
    // cout << " QR Analysis TIME TAKEN " << tt2 - tt1 <<endl;
    
    // hipStreamSynchronize(stream); tt1 = second();

    // printf("step 4: workspace for qr(A)\n");
    checkCudaErrors(cusolverSpDcsrqrBufferInfo(
           cusolverSpH, rowsA, colsA, nnzA,
           descrA, d_csrValB, d_csrRowPtrB, d_csrColIndB,
           d_info,
           &size_internal,
           &size_chol));
    
    //    printf("GPU buffer size = %lld bytes\n", (signed long long)size_chol);
    //    if (buffer_gpu) {
    //        checkCudaErrors(hipFree(buffer_gpu));
    //    }
   checkCudaErrors(hipMalloc(&buffer_gpu, sizeof(char)*size_chol));

   hipStreamSynchronize(stream); tt2 = second(); 
   cout << " QR Buffer setup TIME TAKEN " << tt2 - tt1 <<endl;
   
   const double zero = 0.0;

    hipStreamSynchronize(stream);      tt1 = second(); 

    //    printf("step 5: compute A = L*L^T \n");
   checkCudaErrors(cusolverSpDcsrqrSetup(
           cusolverSpH, rowsA, colsA, nnzA,
           descrA, d_csrValB, d_csrRowPtrB, d_csrColIndB,
           zero,
           d_info));

    hipStreamSynchronize(stream); tt2 = second();
    cout << " QR SETUP TIME TAKEN " << tt2 - tt1 <<endl;

    double t1  =  second();
    checkCudaErrors(cusolverSpDcsrqrFactor(
        cusolverSpH, rowsA, colsA, nnzA,
        NULL, NULL,
        d_info,
        buffer_gpu));
    
    // hipStreamSynchronize(stream);
    // double t2  =  second();
    // cout << " QR FACTORISATION in kernel : " << t2 - t1 <<endl;


    //    hipStreamSynchronize(stream);


    //    printf("step 6: check if the matrix is singular \n");
    //    checkCudaErrors(cusolverSpDcsrqrZeroPivot(
        //    cusolverSpH, d_info, tol, &singularity));

    //    if ( 0 <= singularity){
    //        fprintf(stderr, "Error: A is not invertible, singularity=%d\n", singularity);
    //         exit(0);
    //    }
        

}


void cudaLowLevelQR::lowLevelQRSolve(double* h_b)
{
    
    for(int row = 0 ; row < rowsA ; row++)
        h_Qb[row] = h_b[h_Q[row]];
        
    checkCudaErrors(hipMemcpyAsync(d_Qb , h_Qb  , sizeof(double)*rowsA , hipMemcpyHostToDevice, stream));


    // Solve the System 
    printf("step 7: solve A*x = b \n");
    checkCudaErrors(cusolverSpDcsrqrSolve(
            cusolverSpH, rowsA, colsA, d_Qb, d_z, d_info, buffer_gpu));
     
    //    hipStreamSynchronize(stream);

    // SCatter the Solution Vector based on the Permutation Vector
     
    checkCudaErrors(hipsparseDsctr (cusparseH,
                                    rowsA,d_z,d_Q,d_x,
                                    HIPSPARSE_INDEX_BASE_ZERO));
    
    checkCudaErrors(hipMemcpyAsync(h_b, d_x, sizeof(double)*rowsA, hipMemcpyDeviceToHost,stream));
    
    hipStreamSynchronize(stream);
}


void cudaLowLevelQR::resetLowLevelQRRoutines()
{
    if (cusolverSpH) { checkCudaErrors(hipsolverSpDestroy(cusolverSpH)); }
    if (cusparseH  ) { checkCudaErrors(hipsparseDestroy(cusparseH)); }
    if (stream     ) { checkCudaErrors(hipStreamDestroy(stream)); }
    if (descrA     ) { checkCudaErrors(hipsparseDestroyMatDescr(descrA)); }
   //  if (h_info     ) { checkCudaErrors(cusolverSpDestroyCsrqrInfoHost(h_info)); }
    if (d_info     ) { checkCudaErrors(cusolverSpDestroyCsrqrInfo(d_info)); }


    // Delete HOst Allocated Arrays
    if (h_csrValB  ) { free(h_csrValB); }
    if (h_csrRowPtrB) { free(h_csrRowPtrB); }
    if (h_csrColIndB) { free(h_csrColIndB); }
    if (h_mapBfromA) { free(h_mapBfromA); }
    if (h_x   ) { free(h_x); }
    if (h_Q  ) { free(h_Q); }
    if (h_Qb){ free(h_Qb); }

    // Delete Temp Buffers
    if (buffer_cpu) { free(buffer_cpu); }
    if (buffer_gpu) { checkCudaErrors(hipFree(buffer_gpu)); }

    // Delete all Allocated Arrays - Device
    if (d_csrValB  ) { checkCudaErrors(hipFree(d_csrValB)); }
     if (d_csrRowPtrB) { checkCudaErrors(hipFree(d_csrRowPtrB)); }
     if (d_csrColIndB) { checkCudaErrors(hipFree(d_csrColIndB)); }
     if (d_x) { checkCudaErrors(hipFree(d_x)); }
     if (d_b) { checkCudaErrors(hipFree(d_b)); }
     if (d_Q) { checkCudaErrors(hipFree(d_Q)); }
     if (d_Qb) { checkCudaErrors(hipFree(d_Qb)); }

     if (d_z) { checkCudaErrors(hipFree(d_z)); }

}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

void cudaLowLevelQR_Optimised::initialiseLowLevelCudaQR_optimised()
{

    checkCudaErrors(hipsolverSpCreate(&cusolverSpH));
    checkCudaErrors(hipsparseCreate(&cusparseH));
    checkCudaErrors(hipStreamCreate(&stream));
    checkCudaErrors(hipsolverSpSetStream(cusolverSpH, stream));
    checkCudaErrors(hipsparseSetStream(cusparseH, stream));

    checkCudaErrors(hipsparseCreateMatDescr(&descrA));

    checkCudaErrors(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));


    // ------------- Opaque Data Structure for QR Factorisation -------------------------- //
    checkCudaErrors(cusolverSpCreateCsrqrInfo(&d_info));


    if (baseA) 
    {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));
    }
    else
    {
        checkCudaErrors(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO));
    }


}

void cudaLowLevelQR_Optimised::lowLevelQR_Factorise_optimised(int* h_csrRowPtrA, int* h_csrColIndA, 
                                                    double* h_csrValA, int _rowsA,int _nnzA
                                                    ,const char* reorder)
{

    rowsA = _rowsA;
    colsA = _rowsA;
    nnzA  = _nnzA;

    // double tt1 , tt2;

    // tt1 = second();
    // Allocate Memory on the Host
    h_Qb       = (double*)malloc(sizeof(double)*rowsA);
   //  h_r        = (double*)malloc(sizeof(double)*rowsA);
    h_Q          = (int*   )malloc(sizeof(int)*colsA);
    h_csrRowPtrB = (int*   )malloc(sizeof(int)*(rowsA+1));
    h_csrColIndB = (int*   )malloc(sizeof(int)*nnzA);
    h_csrValB    = (double*)malloc(sizeof(double)*nnzA);
    h_mapBfromA  = (int*   )malloc(sizeof(int)*nnzA);

    // tt2 = second();

    // cout << " Malloc : " << tt2 - tt1  <<endl;


    assert(NULL != h_csrRowPtrB );
    assert(NULL != h_csrColIndB );
    assert(NULL != h_csrValB    );
    assert(NULL != h_mapBfromA  );
    assert(NULL != h_Q  );
    assert(NULL != h_Qb  );

    



    //Allocate Memory on the Device 
     checkCudaErrors(hipMalloc((void **)&d_csrRowPtrB, sizeof(int)*(rowsA+1)));
     checkCudaErrors(hipMalloc((void **)&d_csrColIndB, sizeof(int)*nnzA));
     checkCudaErrors(hipMalloc((void **)&d_csrValB   , sizeof(double)*nnzA));
     checkCudaErrors(hipMalloc((void **)&d_Q, sizeof(int)*colsA));
     checkCudaErrors(hipMalloc((void **)&d_z, sizeof(double)*colsA));
     checkCudaErrors(hipMalloc((void **)&d_x, sizeof(double)*colsA));
     checkCudaErrors(hipMalloc((void **)&d_Qb, sizeof(double)*rowsA));

        // tt1 = second();
     // Reorder the Host Matrix
     if ( 0 == strcmp(reorder, "symrcm") )
     {
        printf(" SYMRCM Reordering Performed ");
         printf("step 2.1: Q = symrcm(A) \n");
         checkCudaErrors(cusolverSpXcsrsymrcmHost(
            cusolverSpH, rowsA, nnzA,
             descrA, h_csrRowPtrA,h_csrColIndA,
             h_Q));
     }
     else if ( 0 == strcmp(reorder, "symamd") )
     {
        printf(" SYMAND Reordering Performed ");
         printf("step 2.1: Q = symamd(A) \n");
         checkCudaErrors(cusolverSpXcsrsymamdHost(
            cusolverSpH, rowsA, nnzA,
             descrA,h_csrRowPtrA,h_csrColIndA,
             h_Q));
     }
     else if ( 0 == strcmp(reorder, "metis") )
     {
         printf("step 2.1: Q = metis(A) \n");
        checkCudaErrors(cusolverSpXcsrmetisndHost(
           cusolverSpH, rowsA, nnzA,
          descrA, h_csrRowPtrA, h_csrColIndA,
            NULL, /* default setting. */
            h_Q)); 

     }
     else 
     {
         fprintf(stderr, "Error: %s is unknown reordering\n", reorder);
         exit(0);
     }


    // Get a current copy of the matrix to array B
    memcpy(h_csrRowPtrB, h_csrRowPtrA, sizeof(int)*(rowsA+1));
    memcpy(h_csrColIndB, h_csrColIndA, sizeof(int)*nnzA);


     /* h_mapBfromA = Identity */
     for(int j = 0 ; j < nnzA ; j++)
     h_mapBfromA[j] = j;
 
    size_t size_perm;
    checkCudaErrors(cusolverSpXcsrperm_bufferSizeHost(
         cusolverSpH, rowsA, colsA, nnzA,
         descrA, h_csrRowPtrB, h_csrColIndB,
         h_Q, h_Q,
         &size_perm));
 
    //  if (buffer_cpu)
    //  {
    //      free(buffer_cpu);
    //  }
     buffer_cpu = (void*)malloc(sizeof(char)*size_perm);
     assert(NULL != buffer_cpu);
 
     // Compute Permutation 
     checkCudaErrors(cusolverSpXcsrpermHost(
         cusolverSpH, rowsA, colsA, nnzA,
         descrA, h_csrRowPtrB, h_csrColIndB,
         h_Q, h_Q,
         h_mapBfromA,
         buffer_cpu));

        // Calculate B = Q*A*Q'
     for(int j = 0 ; j < nnzA ; j++)
     {
         h_csrValB[j] = h_csrValA[ h_mapBfromA[j] ];
     }



    // Get a current copy of the matrix to array B
    checkCudaErrors(hipMemcpyAsync(d_csrRowPtrB, h_csrRowPtrB, sizeof(int)*(rowsA+1), hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_csrColIndB, h_csrColIndB, sizeof(int)*nnzA     , hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_csrValB   , h_csrValB   , sizeof(double)*nnzA  , hipMemcpyHostToDevice, stream));
    checkCudaErrors(hipMemcpyAsync(d_Q         , h_Q         , sizeof(int)*rowsA    , hipMemcpyHostToDevice, stream));



    // printf("step 2: create opaque info structure\n");
    checkCudaErrors(cusolverSpCreateCsrqrInfo(&d_info));

    // hipStreamSynchronize(stream); tt1 = second();
    // printf("step 3: analyze qr(A) to know structure of L\n");
    checkCudaErrors(cusolverSpXcsrqrAnalysis(
                    cusolverSpH, rowsA, colsA, nnzA,
                    descrA, d_csrRowPtrB, d_csrColIndB,
                    d_info));
    
    // printf("step 4: workspace for qr(A)\n");
    checkCudaErrors(cusolverSpDcsrqrBufferInfo(
        cusolverSpH, rowsA, colsA, nnzA,
        descrA, d_csrValB, d_csrRowPtrB, d_csrColIndB,
        d_info,
        &size_internal,
        &size_chol));
      
        checkCudaErrors(hipMalloc(&buffer_gpu, sizeof(char)*size_chol));

        const double zero = 0.0;

        // printf("step 5: compute A = L*L^T \n");
        checkCudaErrors(cusolverSpDcsrqrSetup(
                cusolverSpH, rowsA, colsA, nnzA,
                descrA, d_csrValB, d_csrRowPtrB, d_csrColIndB,
                zero,
                d_info));

        checkCudaErrors(cusolverSpDcsrqrFactor(
            cusolverSpH, rowsA, colsA, nnzA,
            NULL, NULL,
            d_info,
            buffer_gpu));
}

void cudaLowLevelQR_Optimised::lowLevelQR_ReFactorise_Optimised(double* h_valA)
{

    // Allocate  Memory For values Array only
    h_csrValB    = (double*)malloc(sizeof(double)*nnzA);
    assert(NULL != h_csrValB);

    // Map the Values array based on permutation matrix
    for(int j = 0 ; j < nnzA ; j++)
    {
        h_csrValB[j] = h_valA[ h_mapBfromA[j] ];
    }

    // Allocate Memory on GPU
    checkCudaErrors(hipMalloc((void **)&d_csrValB   , sizeof(double)*nnzA));
    checkCudaErrors(hipMalloc((void **)&d_z, sizeof(double)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_x, sizeof(double)*colsA));
    checkCudaErrors(hipMalloc((void **)&d_Qb, sizeof(double)*rowsA));

    checkCudaErrors(hipMemcpyAsync(d_csrValB   , h_csrValB   , sizeof(double)*nnzA  , hipMemcpyHostToDevice, stream));


    checkCudaErrors(hipMalloc(&buffer_gpu, sizeof(char)*size_chol));
    
    double zero = 0.0;
    checkCudaErrors(cusolverSpDcsrqrSetup(
        cusolverSpH, rowsA, colsA, nnzA,
        descrA, d_csrValB, d_csrRowPtrB, d_csrColIndB,
        zero,
        d_info));

    checkCudaErrors(cusolverSpDcsrqrFactor(
        cusolverSpH, rowsA, colsA, nnzA,
        NULL, NULL,
        d_info,
        buffer_gpu));

}



void cudaLowLevelQR_Optimised::lowLevelQRSolve_optimised(double* h_b)
{
    
    for(int row = 0 ; row < rowsA ; row++)
        h_Qb[row] = h_b[h_Q[row]];
        
    checkCudaErrors(hipMemcpyAsync(d_Qb , h_Qb  , sizeof(double)*rowsA , hipMemcpyHostToDevice, stream));


    // Solve the System 
    printf("step 7: solve A*x = b \n");
    checkCudaErrors(cusolverSpDcsrqrSolve(
            cusolverSpH, rowsA, colsA, d_Qb, d_z, d_info, buffer_gpu));
     
    //    hipStreamSynchronize(stream);

    // SCatter the Solution Vector based on the Permutation Vector
     
    checkCudaErrors(hipsparseDsctr (cusparseH,
                                    rowsA,d_z,d_Q,d_x,
                                    HIPSPARSE_INDEX_BASE_ZERO));
    
    checkCudaErrors(hipMemcpyAsync(h_b, d_x, sizeof(double)*rowsA, hipMemcpyDeviceToHost,stream));
    
    hipStreamSynchronize(stream);
}


void cudaLowLevelQR_Optimised::resetLowLevelQR_Refactorise_optimised()
{

    // Delete HOst Allocated Arrays
    if (h_csrValB  ) { free(h_csrValB); }



    // Delete Temp Buffers
    if (buffer_cpu) { free(buffer_cpu); }
    if (buffer_gpu) { checkCudaErrors(hipFree(buffer_gpu)); }

    // Delete all Allocated Arrays - Device
    if (d_csrValB  ) { checkCudaErrors(hipFree(d_csrValB)); }
     if (d_x) { checkCudaErrors(hipFree(d_x)); }
     if (d_Qb) { checkCudaErrors(hipFree(d_Qb)); }
     if (d_z) { checkCudaErrors(hipFree(d_z)); }

}


void cudaLowLevelQR_Optimised::masterResetlowLevelQR_optimised()
{
    if (cusolverSpH) { checkCudaErrors(hipsolverSpDestroy(cusolverSpH)); }
    if (cusparseH  ) { checkCudaErrors(hipsparseDestroy(cusparseH)); }
    if (stream     ) { checkCudaErrors(hipStreamDestroy(stream)); }
    if (descrA     ) { checkCudaErrors(hipsparseDestroyMatDescr(descrA)); }
    if (d_info     ) { checkCudaErrors(cusolverSpDestroyCsrqrInfo(d_info)); }

    // Delete HOst Allocated Arrays
    if (h_csrValB  ) { free(h_csrValB); }
    if (h_csrRowPtrB) { free(h_csrRowPtrB); }
    if (h_csrColIndB) { free(h_csrColIndB); }
    if (h_mapBfromA) { free(h_mapBfromA); }
    // if (h_x   ) { free(h_x); }
    if (h_Q  ) { free(h_Q); }
    if (h_Qb){ free(h_Qb); }

    // Delete Temp Buffers
    // if (buffer_cpu) { free(buffer_cpu); }
    if (buffer_gpu) { checkCudaErrors(hipFree(buffer_gpu)); }



    // Delete all Allocated Arrays - Device
    if (d_csrValB  ) { checkCudaErrors(hipFree(d_csrValB)); }
     if (d_csrRowPtrB) { checkCudaErrors(hipFree(d_csrRowPtrB)); }
     if (d_csrColIndB) { checkCudaErrors(hipFree(d_csrColIndB)); }
     if (d_x) { checkCudaErrors(hipFree(d_x)); }
     if (d_b) { checkCudaErrors(hipFree(d_b)); }
     if (d_Q) { checkCudaErrors(hipFree(d_Q)); }
     if (d_Qb) { checkCudaErrors(hipFree(d_Qb)); }

     if (d_z) { checkCudaErrors(hipFree(d_z)); }

}
